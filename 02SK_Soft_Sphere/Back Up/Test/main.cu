#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 * Created on: 01/06/2016
 * Author: Daniel Varela Varela
 * @: Instituto de Fisica Manuel Sandoval Vallarta UASLP
 * nvcc -arch=sm_35 -rdc=true -lcudadevrt main.cu -o aging
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "scgle.h"
#include <omp.h>

int main(){
  scgle *data;
  FILE *inputOptions;
  double phi,deltaTemp,t, uArresto;
  inputOptions = fopen("options.txt","r");
  data = nuevaSCGLE();
  int i;
  char nom[50];

  if(inputOptions){
    allocateMemorry(&data,inputOptions);
    hipSetDevice(data->device); /* ._. */
    t = data->temI;
    phi = data->phi;
    deltaTemp = (data->temI-data->temF)/data->ts;
    data->dss = blip(t);
    data->phi = phi*pow(data->dss,3);
    printf("Dss: %f, phi:%f\n",data->dss,data->phi);
    data->ski = factorDeEstructura(data,"factor.dat"); /* EL CHIDO */
    sprintf(nom,"%s/skmax%d.dat",date,0);
    skmax = fopen(nom, "w");
    sprintf(nom,"%s/grmax%d.dat",date,0);
    grmax = fopen(nom, "w");
    t-=deltaTemp;
    data->dss = blip(t);//****
    data->phi = phi*pow(data->dss,3);//***
    printf("Dss: %f, phi:%f\n",data->dss,data->phi);
    sprintf(nom,"factor%d.dat",0);
    data->skf = factorDeEstructura(data,nom);
    data->sk = (double*)malloc(sizeof(double)*data->kas);
    data->phi = phi;
    minmaxSk(&data);		/*  */
    uArresto = uDeArresto(data);
    data->deltaU = uArresto/data->us;
    quench(data, 0, 2,0);
    memcpy(data->ski, data->sk, sizeof(double)*data->kas);
    free(data->skf);
    fclose(skmax);
    fclose(grmax);
    fft(data, data->sk);
    for(i = 1; i < data->ts && !data->arrestado; i++, t -= deltaTemp){
      sprintf(nom,"%s/skmax%d.dat",date,i);
      skmax = fopen(nom, "w");
      sprintf(nom,"%s/grmax%d.dat",date,0);
      grmax = fopen(nom, "w");
      data->dss = blip(t-deltaTemp);//*****
      data->phi = phi*pow(data->dss,3);//*****
      printf("Dss: %f, phi:%f\n",data->dss,data->phi);
      sprintf(nom,"factor%d.dat",i);
      data->skf = factorDeEstructura(data,nom);
      data->phi = phi;
      uArresto = uDeArresto(data);
      data->deltaU = uArresto/data->us;
      quench(data, 0, uArresto,i);
      free(data->ski);
      data->ski = data->sk;
      data->sk = (double*)malloc(sizeof(double)*data->kas);
      free(data->skf);
      fclose(skmax);
      fclose(grmax);
    }
    fclose(efeself);
    printf("Fin del Programa\n");
  }

  return 0;
}

